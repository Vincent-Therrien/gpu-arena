#include <iostream>
#include <hip/hip_runtime.h>

#define N 1024  // Array size (must be a power of 2 for reduction)
#define THREADS_PER_BLOCK 256

// CUDA kernel to sum array elements using parallel reduction
__global__ void sumReduction(float *input, float *output) {
    __shared__ float sharedData[THREADS_PER_BLOCK];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Load elements into shared memory
    sharedData[tid] = (idx < N) ? input[idx] : 0.0f;
    __syncthreads();

    // Perform parallel reduction
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sharedData[tid] += sharedData[tid + s];
        }
        __syncthreads();
    }

    // Store result from each block
    if (tid == 0) {
        output[blockIdx.x] = sharedData[0];
    }
}

// Host function to launch kernel
float sumArrayOnGPU(float *h_array) {
    float *d_array, *d_partialSums;
    int numBlocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    hipMalloc(&d_array, N * sizeof(float));
    hipMalloc(&d_partialSums, numBlocks * sizeof(float));

    hipMemcpy(d_array, h_array, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    sumReduction<<<numBlocks, THREADS_PER_BLOCK>>>(d_array, d_partialSums);

    // Copy partial sums back to host
    float *h_partialSums = new float[numBlocks];
    hipMemcpy(h_partialSums, d_partialSums, numBlocks * sizeof(float), hipMemcpyDeviceToHost);

    // Final sum on CPU
    float totalSum = 0.0f;
    for (int i = 0; i < numBlocks; i++) {
        totalSum += h_partialSums[i];
    }

    // Cleanup
    hipFree(d_array);
    hipFree(d_partialSums);
    delete[] h_partialSums;

    return totalSum;
}

// CPU code
int main() {
    float h_array[N];
    for (int i = 0; i < N; i++) {
        h_array[i] = 1.0f;
    }
    float sum = sumArrayOnGPU(h_array);
    std::cout << "Sum: " << sum << std::endl;
    return 0;
}
